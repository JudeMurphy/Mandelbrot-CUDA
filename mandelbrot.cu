#include "hip/hip_runtime.h"
//JUDE MURPHY
//PARALLEL AND SCIENTIFIC COMPUTING
//ASSIGNMENT 2

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define N 25000000

__global__ void plotMandelbrotSet(int width, int height, double xcenter, double ycenter, double resolution, double gamma, int max_iter, double *matrixR, double *matrixG, double *matrixB);

int main(int argc, char** argv)
{	
	//ALLOCATE ALL VARIABLES
	int height, width;
	double xcenter, ycenter;
	double resolution;
	double gamma;
	int max_iter;
	FILE *outfp;

	//GET ARGUMENTS FROM RUNTIME ARGS
	height = atoi(argv[1]);
	width = atoi(argv[2]);
	xcenter = atof(argv[3]);
	ycenter = atof(argv[4]);
	resolution = atof(argv[5]);
	gamma = atof(argv[6]);
	max_iter = atoi(argv[7]);
	outfp = fopen(argv[8], "w");

	//MAKE TOTAL GRID SPACE BE FROM THE HEIGHT TIMES THE WIDTH
	int totalGridSpace = height * width;

	//USED TO DETERMINE THE TIME TAKEN TO COMPLETE THE PROGRAM
	hipEvent_t cudaStart, cudaEnd;
	float elapsedTime;

	//NEED FOR TIME TO START
	srand(time(NULL));

	//_________________________________________________________________________
	//ALLOCATES MEMORY FOR THREE ARRAYS ON THE CPU	
	double * matrixR = (double *) malloc(totalGridSpace * sizeof(double));
	double * matrixG = (double *) malloc(totalGridSpace * sizeof(double));
	double * matrixB = (double *) malloc(totalGridSpace * sizeof(double));

	//_________________________________________________________________________
	//CREATE CUDA EVENTS AND START RECORDING TIME
	hipEventCreate(&cudaStart);
	hipEventCreate(&cudaEnd);
	hipEventRecord(cudaStart, 0);

	//_________________________________________________________________________
	//ALLOCATES MEMORY FOR THREE ARRAYS ON THE GPU
	double * dev_R, *dev_G, *dev_B;
	hipMalloc((void**)&dev_R, totalGridSpace * sizeof(double));
	hipMalloc((void**)&dev_G, totalGridSpace * sizeof(double));
	hipMalloc((void**)&dev_B, totalGridSpace * sizeof(double));

	//COMPUTE MANDELBROT SET IN PARALLEL
	plotMandelbrotSet<<<256, 256>>>(width, height, xcenter, ycenter, resolution, gamma, max_iter, dev_R, dev_G, dev_B);

	//COPY DATA BACK TO CPU AFTER FINISHING COMPUTATION
	hipMemcpy(matrixR, dev_R, (totalGridSpace * sizeof(double)), hipMemcpyDeviceToHost);
	hipMemcpy(matrixG, dev_G, (totalGridSpace * sizeof(double)), hipMemcpyDeviceToHost);
	hipMemcpy(matrixB, dev_B, (totalGridSpace * sizeof(double)), hipMemcpyDeviceToHost);

	//GET THE END OF THE TIME IT TOOK TO CALCULATE
	hipEventRecord(cudaEnd, 0);
	hipEventSynchronize(cudaEnd);	
	hipEventElapsedTime(&elapsedTime, cudaStart, cudaEnd);

	//CLOSE OUT EVENTS
	hipEventDestroy(cudaStart);
	hipEventDestroy(cudaEnd);

	printf("CUDA Elapsed Time: %3.3f sec\n", elapsedTime/1000);

	//WRITING OUT TO THE FILE
	for (int i = 0; i < totalGridSpace; i++)
	{
		fprintf(outfp, " %lf %lf %lf\n", matrixR[i], matrixG[i], matrixB[i]);
	}

	//FREE MEMORY
	hipFree(dev_R);
	hipFree(dev_G);
	hipFree(dev_B);
	free(matrixR);
	free(matrixG);
	free(matrixB);	

	//RESET CUDA DEVICE
	hipDeviceReset();

	return 0;
}

__global__ void plotMandelbrotSet(int width, int height, double xcenter, double ycenter, double resolution, double gamma, int max_iter, double *matrixR, double *matrixG, double *matrixB)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int totalNumberOfBoxes = height * width;

	while (id < totalNumberOfBoxes)
	{
		int c = id / width;
		int r = id % width;
		
		int currentIndex = c + (r * width);

		double xoffset = -(width-1)/2.0;
		double yoffset = (height-1)/2.0;
		
		double x = xcenter + (xoffset + c)/resolution;
		double y = ycenter + (yoffset - r)/resolution;

		int iter = 0;
		double a = 0.0, b = 0.0, a_old = 0.0, b_old = 0.0;
		double dist_sqr = 0.0;
		
		while (iter<max_iter && dist_sqr<=4.0)
		{
			iter++;
			a = a_old*a_old - b_old*b_old + x;
			b = 2.0*a_old*b_old + y;
			dist_sqr = a*a + b*b;
			a_old = a;
			b_old = b;
		}
		if (iter == max_iter)
		{
			matrixR[currentIndex] = 0.0f;
			matrixG[currentIndex] = 0;
			matrixB[currentIndex] = 0;
		}
		else
		{
			matrixR[currentIndex] = (double)pow(((double) iter)/((double)max_iter), gamma);
			matrixG[currentIndex] = 1.0;
			matrixB[currentIndex] = 1.0;
		}

		id += blockDim.x * gridDim.x;
	}	
}
